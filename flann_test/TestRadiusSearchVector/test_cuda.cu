#include "hip/hip_runtime.h"
#include <time.h>
#define FLANN_USE_CUDA
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <vector_functions.h>
#include "../include/test_cuda.h"
#include <flann/util/matrix.h>

#include <stdlib.h>
#include <stdio.h>
#include <flann/flann.h>

extern "C" void wrapper_kernel_radius(float* point_arr,int *a,flann_index_t idx_ptr) {

    float r =0.25;
    int n_points=50000;
    int D=3;

    Matrix<int> m_indices(a, 50000, 1);
    flann::Matrix<float> query = flann:: Matrix<float>(point_arr, n_points, D);
    flann::Matrix<float> dists= flann::Matrix<float>(new float[n_points*1], n_points, 1);
    flann::KDTreeCuda3dIndex<L2_Simple<float>>* index =(flann::KDTreeCuda3dIndex<L2_Simple<float>>*)idx_ptr;
    
    thrust::host_vector<float4> data_host(n_points);
    for (int i = 0; i < query.rows; i++)
    {

        data_host[i] = make_float4(query[i][0], query[i][1], query[i][2], 0);
    }
    thrust::device_vector<float4> data_device = data_host;
    float4 b = data_device[2];
    std::cout<<"data:" << b.x << " "<< b.y << " " << b.z << " "<< b.w<< std::endl;
    
    
    
    
    
    index->radiusSearch(query, m_indices,dists, r*r, flann::SearchParams(-1) ); 
    return;
}
extern "C" flann_index_t wrapper_build_tree(float* point_arr) {
    int n_points=50000;
    int D=3;
    flann::Matrix<float> query = flann:: Matrix<float>(point_arr, n_points, D);
    thrust::host_vector<float4> query_host(query.rows);
    for( int i=0; i<query.rows; i++ )
	{
		query_host[i]=make_float4(query[i][0],query[i][1],query[i][2],0);
	}
    thrust::device_vector<float4> query_device = query_host;

    flann::Matrix<float> query_device_matrix( (float*)thrust::raw_pointer_cast(&query_device[0]),query.rows,3,4*4);
    flann::KDTreeCuda3dIndexParams index_params;
    index_params["input_is_gpu_float4"] = true;
   
    flann::KDTreeCuda3dIndex<L2<float> >* idx = new KDTreeCuda3dIndex<L2<float> >(query_device_matrix, index_params);
   
    idx->buildIndex();

    return (flann_index_t) idx ;

}
 