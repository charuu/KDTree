#include "hip/hip_runtime.h"

#include <time.h>
#define FLANN_USE_CUDA


#include <flann/util/matrix.h>
#include <vector>
#include <set>

#include<stdlib.h>
#include<stdio.h>
#include <flann/flann.h>
#include <flann/io/hdf5.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <vector_functions.h>

 void start_timer(const std::string& message = "")
    {
        clock_t start_time_;
        if (!message.empty()) {
            printf("%s", message.c_str());
            fflush(stdout);
        }
        start_time_ = clock();
    }

    double stop_timer()
    {
        clock_t start_time_;
        return double(clock()-start_time_)/CLOCKS_PER_SEC;
    }
int find_correspondences(){
    printf("Reading test data...");
    fflush(stdout);
    int nn=3;
    flann::Matrix<float> data;
    flann::Matrix<float> query;
    flann::Matrix<float> dists;
    flann::Matrix<int> indices;
    Matrix<int> match(new int[query.rows*nn], data.rows, nn);
    
    const int n_points=50000;
    printf("creating random point cloud (%d points)...", n_points);
    data = flann::Matrix<float>(new float[n_points*3], n_points,3);
    srand(1);
    FILE *fp = fopen("datasets/XX_after.txt", "r");
    int RANGE_MAX = 100;
    const int max_nn = 1;
		
    flann::Matrix<float> gt_dists;
    gt_dists = flann::Matrix<float>(new float[query.rows*max_nn], query.rows, max_nn);
   
    if (fp == NULL)
    {
        puts("Couldn't open file");
        exit(0);
    }
    int n =50000;
    int dim=3;
    char line[120];
    char* end;
    char* token;
    float val;
    float point_arr[50000][3];
    int i =0;
    
    while(fgets(line,120,fp)){
        int j=0;
        token  = strtok(line, "\t");
        while(token!=NULL){
            val = strtod(token,&end);
            point_arr[i][j]=val;
            //printf("%lf \t",val);
            token = strtok(NULL, "\t");
            j++;
        }
        i++;
      //   printf("\n");
    }
    
    for( int i=0; i<n_points; i++ )
    {
        data[i][0]=point_arr[i][0];//rand()/float(RAND_MAX);
        data[i][1]=point_arr[i][1];//rand()/float(RAND_MAX);
        data[i][2]=point_arr[i][2];//rand()/float(RAND_MAX);
        //   std::cout<<data[i][0]<<" "<<data[i][1]<<" "<<data[i][2]<<std::endl;
    }
    query = flann::Matrix<float>(new float[n_points*3], n_points,3);
    for( int i=0; i<n_points; i++ )
    {
        query[i][0]=point_arr[i][0];
        query[i][1]=point_arr[i][1];
        query[i][2]=point_arr[i][2];
        // std::cout<<data[i][0]<<" "<<data[i][1]<<" "<<data[i][2]<<std::endl;
    }
    thrust::host_vector<float4> data_host(data.rows);
	for( int i=0; i<data.rows; i++ )
	{
		data_host[i]=make_float4(data[i][0],data[i][1],data[i][2],0);
	}
	thrust::device_vector<float4> data_device = data_host;
	thrust::host_vector<float4> query_host(data.rows);
	for( int i=0; i<data.rows; i++ )
	{
		query_host[i]=make_float4(query[i][0],query[i][1],query[i][2],0);
	}
	thrust::device_vector<float4> query_device = query_host;
	
	flann::Matrix<float> data_device_matrix( (float*)thrust::raw_pointer_cast(&data_device[0]),data.rows,3,4*4);
	flann::Matrix<float> query_device_matrix( (float*)thrust::raw_pointer_cast(&query_device[0]),data.rows,3,4*4);
	
	flann::KDTreeCuda3dIndexParams index_params;
	index_params["input_is_gpu_float4"]=true;
	flann::KDTreeCuda3dIndex<L2_Simple<float> > index(data_device_matrix, index_params);
    start_timer("Building kd-tree index...");
    index.buildIndex();
    printf("done (%g seconds)\n", stop_timer());

	
	thrust::device_vector<int> indices_device(query.rows*4);
	thrust::device_vector<float> dists_device(query.rows*4);
	flann::Matrix<int> indices_device_matrix( (int*)thrust::raw_pointer_cast(&indices_device[0]),query.rows,4);
	flann::Matrix<float> dists_device_matrix( (float*)thrust::raw_pointer_cast(&dists_device[0]),query.rows,4);
	
    start_timer("Searching KNN...");
	indices.cols=4;
	dists.cols=4;
	flann::SearchParams sp;
	sp.matrices_in_gpu_ram=true;
    index.knnSearch(query_device_matrix, indices_device_matrix, dists_device_matrix, 4, sp );
    printf("done (%g seconds)\n", stop_timer());
	
	flann::Matrix<int> indices_host( new int[ query.rows*4],query.rows,4 );
	flann::Matrix<float> dists_host( new float[ query.rows*4],query.rows,4 );
	
	thrust::copy( dists_device.begin(), dists_device.end(), dists_host.ptr() );
	thrust::copy( indices_device.begin(), indices_device.end(), indices_host.ptr() );

    // float precision = computePrecisionDiscrete(gt_dists,dists_host, 1e-08);

    //printf("Precision: %g\n", precision);
	fclose(fp);
	delete [] indices_host.ptr();
	delete [] dists_host.ptr();
}

extern "C" void wrapper_kernel() {
     find_correspondences();
     return;
}